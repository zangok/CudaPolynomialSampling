#include "hip/hip_runtime.h"
#include "PolynomialSampling.cuh"
#include <iostream>
#include <chrono>
#include "CountPositives.cuh"
#include "utils.cuh"
#include <hip/hip_runtime.h>
#include ""
#include <cmath>


//A atomic each thread does a count, may be unefficent.
//Another implementation I did think of was to use y as the storage for counting positives?
//I found that alot of the times, a atomic add was significantly faster than the reduction method below in
//Nvidia NSight
__global__ void runPositivesKernel(int* count, const double* __restrict__ y, int N) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Use a grid-stride loop to ensure all elements are checked
	for (int i = tid; i < N; i += gridDim.x * blockDim.x) {
		if (y[i] > 0) {
			atomicAdd(count, 1);
		}
	}
}

__global__ void runPositivesKernel_branchless(int* d_global_count, const double* __restrict__ y, int N) {
	// Shared memory array to store the counts for the current block
	extern __shared__ int s_block_count[];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int local_count = 0;

	// Use a grid-stride loop to ensure all elements are checked
	for (int i = tid; i < N; i += gridDim.x * blockDim.x) {
		// This is the key change. The boolean expression (y[i] > 0)
		// evaluates to 1 or 0, which we can directly add without a conditional branch.
		local_count += (y[i] > 0);
	}

	// Store the local sum in shared memory
	s_block_count[threadIdx.x] = local_count;
	__syncthreads();

	
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			s_block_count[threadIdx.x] += s_block_count[threadIdx.x + s];
		}
		__syncthreads();
	}

	// The first thread of the block atomically updates the global counter
	if (threadIdx.x == 0) {
		atomicAdd(d_global_count, s_block_count[0]);
	}
}

// Calculates positive count using the device array y
int calc_positives(double* d_y, const SamplingRange& h_range_in) {
	int N = h_range_in.count;

	if (N <= 0) return 0;

	int* d_count = nullptr;
	int h_count = 0; // Correctly initialize host variable
	CUDA_CHECK(hipMalloc(&d_count, sizeof(int)));
	CUDA_CHECK(hipMemset(d_count, 0, sizeof(int)));

	//Launch config
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	// Pass the array size N d_y to the kernel for the grid-stride loop
	//runPositivesKernel << <blocksPerGrid, threadsPerBlock >> > (d_count, d_y, N);
	runPositivesKernel_branchless << <blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int) >> > (d_count, d_y, N);

	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());
	
	//copy the result from the device to the host
	CUDA_CHECK(hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost));

	hipFree(d_count);

	return h_count;
}
