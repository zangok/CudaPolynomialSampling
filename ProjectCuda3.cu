
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>


/*
Idea: All weights abd bias in global mem
Divide n samples equally on GPU threads.
keep array of values on device for reduced transfers
This minimizes transfers.
*/

#include "PolynomialSampling.cuh" 
#include "utils.cuh"
#include "ExperimentManager.cuh"


int main() {
    // 1. Define and initialize host data for the experiment
    double h_coeffs[] = {1.0, 0.5, -1.0, 0.15};
    int degree = 3;
    int sample_count = 10000000;

    Polynomial h_poly{};
    for (int i = 0; i <= degree; ++i) {
        h_poly.coeffs[i] = h_coeffs[i];
    }
    h_poly.degree = degree;

    SamplingRange h_range = { -10.0, 20.0/sample_count, sample_count +1};

    // 2. Call the simple experiment runner function
    runExperiment(h_poly, h_range);

    std::cin.get();

    return 0;
}